
#include <hip/hip_runtime.h>
#include <stdint.h>

__constant__ unsigned char target[32];

__constant__ unsigned char nbits[4];
__constant__ unsigned char ntime[4];
__constant__ unsigned char version[4];
__constant__ unsigned char previous_block_hash[32];

__constant__ int len_extranonce2;
__constant__ int len_prefix_coinbase;
__constant__ int len_suffix_coinbase;
__constant__ int merkle_branch_depth;

__device__ uint32_t result_nonce;


#define ROTRIGHT(word, bits) (((word) >> (bits)) | ((word) << (32 - (bits))))
#define CH(x, y, z) (((x) & (y)) ^ (~(x) & (z)))
#define MAJ(x, y, z) (((x) & (y)) ^ ((x) & (z)) ^ ((y) & (z)))
#define EP0(x) (ROTRIGHT(x, 2) ^ ROTRIGHT(x, 13) ^ ROTRIGHT(x, 22))
#define EP1(x) (ROTRIGHT(x, 6) ^ ROTRIGHT(x, 11) ^ ROTRIGHT(x, 25))
#define SIG0(x) (ROTRIGHT(x, 7) ^ ROTRIGHT(x, 18) ^ ((x) >> 3))
#define SIG1(x) (ROTRIGHT(x, 17) ^ ROTRIGHT(x, 19) ^ ((x) >> 10))



__constant__ unsigned int k[64] = {
    0x428a2f98, 0x71374491, 0xb5c0fbcf, 0xe9b5dba5, 0x3956c25b, 0x59f111f1, 0x923f82a4, 0xab1c5ed5,
    0xd807aa98, 0x12835b01, 0x243185be, 0x550c7dc3, 0x72be5d74, 0x80deb1fe, 0x9bdc06a7, 0xc19bf174,
    0xe49b69c1, 0xefbe4786, 0x0fc19dc6, 0x240ca1cc, 0x2de92c6f, 0x4a7484aa, 0x5cb0a9dc, 0x76f988da,
    0x983e5152, 0xa831c66d, 0xb00327c8, 0xbf597fc7, 0xc6e00bf3, 0xd5a79147, 0x06ca6351, 0x14292967,
    0x27b70a85, 0x2e1b2138, 0x4d2c6dfc, 0x53380d13, 0x650a7354, 0x766a0abb, 0x81c2c92e, 0x92722c85,
    0xa2bfe8a1, 0xa81a664b, 0xc24b8b70, 0xc76c51a3, 0xd192e819, 0xd6990624, 0xf40e3585, 0x106aa070,
    0x19a4c116, 0x1e376c08, 0x2748774c, 0x34b0bcb5, 0x391c0cb3, 0x4ed8aa4a, 0x5b9cca4f, 0x682e6ff3,
    0x748f82ee, 0x78a5636f, 0x84c87814, 0x8cc70208, 0x90befffa, 0xa4506ceb, 0xbef9a3f7, 0xc67178f2
};

__device__ void sha256_transform(unsigned int *state, const unsigned char *data) {
    unsigned int a, b, c, d, e, f, g, h, t1, t2;//, m[64];
    unsigned int w[64];

    a = state[0];
    b = state[1];
    c = state[2];
    d = state[3];
    e = state[4];
    f = state[5];
    g = state[6];
    h = state[7];

    for (int i = 0; i < 16; i++) {
        w[i] = (data[i * 4] << 24) | (data[i * 4 + 1] << 16) | (data[i * 4 + 2] << 8) | data[i * 4 + 3];
    }

    for (int i = 16; i < 64; i++) {
        w[i] = SIG1(w[i - 2]) + w[i - 7] + SIG0(w[i - 15]) + w[i - 16];
    }

    for (int i = 0; i < 64; i++) {
        t1 = h + EP1(e) + CH(e, f, g) + k[i] + w[i];
        t2 = EP0(a) + MAJ(a, b, c);
        h = g;
        g = f;
        f = e;
        e = d + t1;
        d = c;
        c = b;
        b = a;
        a = t1 + t2;
    }

    state[0] += a;
    state[1] += b;
    state[2] += c;
    state[3] += d;
    state[4] += e;
    state[5] += f;
    state[6] += g;
    state[7] += h;
}

__device__ void sha256(const unsigned char *message, int length, unsigned char *output) {
    unsigned int state[8] = {
        0x6a09e667, 0xbb67ae85, 0x3c6ef372, 0xa54ff53a,
        0x510e527f, 0x9b05688c, 0x1f83d9ab, 0x5be0cd19
    };

    int padded_length = length + 9;
    while (padded_length % 64 != 0) padded_length++;
    unsigned char padded_message[448];
    memcpy(padded_message, message, length);

    padded_message[length] = 0x80;
    for (int i = length + 1; i < padded_length - 8; i++) {
        padded_message[i] = 0;
    }

    unsigned long long bit_length = length * 8;
    for (int i = 0; i < 8; i++) {
        padded_message[padded_length - 8 + i] = (bit_length >> (56 - i * 8)) & 0xff;
    }

    for (int i = 0; i < padded_length; i += 64) {
        sha256_transform(state, padded_message + i);
    }
    for (int i = 0; i < 8; i++) {
        output[i * 4] = (state[i] >> 24) & 0xff;
        output[i * 4 + 1] = (state[i] >> 16) & 0xff;
        output[i * 4 + 2] = (state[i] >> 8) & 0xff;
        output[i * 4 + 3] = state[i] & 0xff;
    }
}

// sha256 twice
__device__ void double_sha256(const unsigned char *_input, int len, unsigned char *_output) {
    unsigned char __hash1[32];
    sha256(_input, len, __hash1);
    sha256(__hash1, 32, _output);
}

// generate random extranonce
__device__ void generate_random_extranonce(unsigned char *_output) {
    //int __tid = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int __seed;
    for(int i = 0; i < len_extranonce2; i++){
        __seed = clock64() ^ (threadIdx.x * 0x96d6) ^ (blockIdx.x * 0xf6b6);
        //_output[__tid * len_extranonce2 + i] = (uint8_t)((__seed + (blockIdx.x * blockDim.x + threadIdx.x)) % 255);
        _output[i] = (unsigned char)((__seed + (blockIdx.x * blockDim.x + threadIdx.x)) % 255);
    }
}

// generate random nonce
__device__ uint32_t generate_random_nonce()
{
    uint32_t __nonce = (clock64() ^ (threadIdx.x * 0x96d6) ^ (blockIdx.x * 0xf6b6)) & 0xFFFFFFFF;
    return __nonce;
}

// create merkle root
__device__ void build_merkle_root(unsigned char *_coinbaseId, unsigned char *_merkle_branch) {
    unsigned char __temp[32];
    memcpy(__temp, _coinbaseId, 32);

    for (int i = 0; i < merkle_branch_depth; i +=32) {
        unsigned char __combined[64];
        memcpy(__combined, __temp, 32);
        memcpy(__combined + 32, _merkle_branch + i, 32);
        double_sha256(__combined, 64, __temp);
    }
    memcpy(_coinbaseId, __temp, 32);
}

// create pre header 76 bytes
__device__ void build_pre_header(unsigned char *_merkle_root, unsigned char *_header) {
    memset(_header, 0, 80);
    memcpy(_header, version, 4);
    memcpy(_header + 4, previous_block_hash, 32);
    memcpy(_header + 36, _merkle_root, 32);
    memcpy(_header + 68, ntime, 4);
    memcpy(_header + 72, nbits, 4);
}

__device__ void reverse_bytes(unsigned char *arr, int size) {
    for (int i = 0; i < size / 2; i++) {
        unsigned char temp = arr[i];
        arr[i] = arr[size - 1 - i];
        arr[size - 1 - i] = temp;
    }
}

//check target and hash
__device__ bool check_target(unsigned char *_header_hash) {
    reverse_bytes(_header_hash, 32);
    for (int i = 0; i < 32; i++)
    {
        if (_header_hash[i] < target[i]) return true;
        if (_header_hash[i] > target[i]) return false;
    }
    return false;
}

__global__ void mine_kernel(unsigned char * _prefix_coinbase, unsigned char * _suffix_coinbase,
                            unsigned char *_merkle_branch, unsigned char *_extranonce2) {

    int __len_coinbase = len_prefix_coinbase + len_extranonce2 + len_suffix_coinbase;
    unsigned char __extranonce2_buffer[32];
    unsigned char __coinbase_buffer[200];

    unsigned char *__extranonce2 = __extranonce2_buffer;
    unsigned char *__coinbase = __coinbase_buffer;

    unsigned char __coinbaseID[32];

    unsigned char __header[80];
    unsigned char __header_hash[32];

    uint32_t __nonce;

    int __extranonce2_loop_counter = 0;
    const int __extranonce2_loop_limit = 8;

    int __nonce_counter = 0;
    //const uint32_t __nonce_limit = 77164; // for 50% chance of duplicate
    //const uint32_t __nonce_limit = 20990; // for 5% chance of duplicate
    //const uint32_t __nonce_limit = 9299; // for 1% chance of duplicate
    const int __nonce_limit = 6554; // for 0.5% chance of duplicate

    while (__extranonce2_loop_counter <= __extranonce2_loop_limit && result_nonce == 0xFFFFFFFF){
        generate_random_extranonce(__extranonce2);

        // build coinbase
        memcpy(__coinbase , _prefix_coinbase, len_prefix_coinbase);
        memcpy(__coinbase + len_prefix_coinbase, __extranonce2, len_extranonce2);
        memcpy(__coinbase + len_prefix_coinbase + len_extranonce2, _suffix_coinbase, len_suffix_coinbase);


        // create coinbase transaction ID
        double_sha256(__coinbase, __len_coinbase, __coinbaseID);

        build_merkle_root(__coinbaseID, _merkle_branch);

        build_pre_header(__coinbaseID, __header);
        __nonce_counter = 0;
        // false if one thread find nonce valid_nonce = 0xFFFFFFFF;
        while (__nonce_counter <= __nonce_limit && result_nonce == 0xFFFFFFFF){
            __nonce = generate_random_nonce();

            // add nonce to header
            __header[76] = (__nonce >> 24) & 0xFF;
            __header[77] = (__nonce >> 16) & 0xFF;
            __header[78] = (__nonce >> 8) & 0xFF;
            __header[79] = __nonce & 0xFF;
            double_sha256(__header, 80, __header_hash);
            if (check_target(__header_hash))
            {
                atomicExch(&result_nonce, __nonce);
                for (int i = 0; i < len_extranonce2; i++){
                    _extranonce2[i] = __extranonce2[i];
                }
            }
            __nonce_counter += 1;
        } // end of __nonce loop
        __extranonce2_loop_counter += 1;
    } // end og __extranonce2 loop
} // end of kernel
